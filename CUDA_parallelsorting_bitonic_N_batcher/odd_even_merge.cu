
#include <hip/hip_runtime.h>
////12163291 강현지 알고리즘설계 HW1
//#pragma warning(disable: 4819) //경고 끄기
//
//#include<stdio.h>
//#include<iostream>
//#include <cuda_runtime.h>
//#include <cuda.h>
//#include <time.h> //for난수
//#include <math.h>
//
//using namespace std;
//
//#define DATASIZE 1048576 //2048 131072 262144 1048576원래 안됐는데 스택 오버플로우 설정 따로 해주니 됐음
//#define BLOCK_SIZE 2048  //1024 2048 2048 2048
////524288 1024 오래걸려서 그렇지 되긴함 2097152 개 //80초,,
////1048576 1024 아무리기다려도..
////1048576 2048 4194304개 0.21초 0.204초 등듣등
////1048576 4089 0.211초등등 우선 2048이랑 별차이는 없음
//
//void print_elapsed(clock_t start, clock_t stop) //시간출력용
//{
//    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
//    printf("CUDA ODD EVEN MERGE\n소요시간 : %.3fs\n", elapsed);
//}
//
//__global__ void oddevensort(int* input, unsigned int len, int i)
//{
//    //개별 블럭의 좌표
//    unsigned int tx = threadIdx.x;
//
//    //전체 이미지의 좌표
//    unsigned int x = tx + blockDim.x * blockIdx.x;
//    //이동에 쓸 임시 변수
//    int temp;
//
//    //자료의 길이만큼 돌리는데, 인덱스(i)가 짝수이면 데이터의 짝수자리와 그 다음 숫자를 비교.
//    //인덱스가 홀수이면 데이터의 홀수자리와 그 다음 숫자를 비교해서 정렬한다.
//    if (i % 2 == 0)
//    {
//        // 길이를 측정안해주면 블럭에 남아있던 자리에 있는 자료가 튀어나올 수 있으니 조심.
//        if (input[x] > input[x + 1] && x < len && x % 2 == 0)
//        {
//            temp = input[x + 1];
//            input[x + 1] = input[x];
//            input[x] = temp;
//        }
//    }
//    else
//    {
//        if (input[x] > input[x + 1] && x < len && x % 2 != 0)
//        {
//            temp = input[x + 1];
//            input[x + 1] = input[x];
//            input[x] = temp;
//        }
//    }
//    __syncthreads();
//}
//
//void array_print(int* arr, int length)
//{
//    int i;
//    for (i = 0; i < length; ++i) {
//        printf("%d ", arr[i]);
//    }
//    printf("\n");
//}
//
//void array_fill(int* arr, int length)
//{
//    srand((int)time(NULL));
//
//    for (int i = 0; i < length; ++i) {
//        arr[i] = rand() % 100;
//    }
//}
//
//__global__ void bitonic_sort_step(int* dev_values, int j, int k)
//{
//    unsigned int i, ixj; /* Sorting partners: i and ixj */
//    i = threadIdx.x + blockDim.x * blockIdx.x;
//    ixj = i ^ j;
//
//    /* The threads with the lowest ids sort the array. */
//    if ((ixj) > i) {
//        if ((i & k) == 0) {
//            /* Sort ascending */
//            if (dev_values[i] > dev_values[ixj]) {
//                /* exchange(i,ixj); */
//                int temp = dev_values[i];
//                dev_values[i] = dev_values[ixj];
//                dev_values[ixj] = temp;
//            }
//        }
//        if ((i & k) != 0) {
//            /* Sort descending */
//            if (dev_values[i] < dev_values[ixj]) {
//                /* exchange(i,ixj); */
//                int temp = dev_values[i];
//                dev_values[i] = dev_values[ixj];
//                dev_values[ixj] = temp;
//            }
//        }
//    }
//}
//
//clock_t start, stop = 0;
//int TestInput[DATASIZE], TestOutput[DATASIZE];
//
//int main(){
//    cout << "12163291\n강현지\n";
//
//    srand(time(NULL));
//
//    for (int i = 0; i < DATASIZE; i++) {
//        TestInput[i] = rand() % 100; //난수생성해줍니다.
//    }
//
//    //device 설정
//    int* devInput, * devOutput;
//    //일단 크기는 아니까
//    unsigned int MemDataSize = DATASIZE * sizeof(int);
//
//    // device 자리 잡아줍니다.
//    cudaMalloc((void**)&devInput, MemDataSize);
//    cudaMalloc((void**)&devOutput, MemDataSize);
//    cudaMemset(devOutput, 0, MemDataSize);
//
//    // 자리 잡았으면 복사를 해줍니다.00000
//    cudaMemcpy(devInput, TestInput, MemDataSize, cudaMemcpyHostToDevice);
//
//    // block 크기 설정합니다.
//    dim3    dimBlocksize(BLOCK_SIZE);
//    dim3    dimGridsize(ceil((DATASIZE - 1) / (float)BLOCK_SIZE) + 1);
//    // 일단 Max값과 min값을 알아야 합니다.
//    // 비교적 간단하니 cpu에게 맡겨줍니다.
//    
//    start = clock();
//    for (int i = 0; i < DATASIZE; i++) {
//        oddevensort << < dimGridsize, dimBlocksize >> > (devInput, DATASIZE, i);
//    }
//    stop = clock();
//
//    // 결과물 testoutput에 복사합니다.
//    cudaMemcpy(TestOutput, devInput, MemDataSize, cudaMemcpyDeviceToHost);
//
// /*   for (int i = 0; i < DATASIZE; i++) {
//        cout << TestOutput[i] << ", ";  //결과 확인용
//        if ((i + 1) % 16 == 0)  {
//            cout << endl;
//        }
//    }*/
//    cout << endl << "배열 크기 : " << sizeof(TestOutput) << endl << endl;
//   print_elapsed(start, stop); //걸린시간 출력해줍니다.
//
//   //위에 GPU에 마련한 자리 free.
//   //malloc - free는 항상 짝이지요. 
//   //항상해줘야 합니다. 
//    cudaFree(devInput);
//    cudaFree(devOutput);
//
//    return 0;
//}
//
////https://github.com/ptjoker95/OddEvenSort-with-CUDA/blob/master/oddevensort.cu
////출처는 위와 같습니다.