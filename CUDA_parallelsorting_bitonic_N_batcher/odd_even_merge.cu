
#include <hip/hip_runtime.h>
////12163291 ������ �˰��򼳰� HW1
//#pragma warning(disable: 4819) //��� ����
//
//#include<stdio.h>
//#include<iostream>
//#include <cuda_runtime.h>
//#include <cuda.h>
//#include <time.h> //for����
//#include <math.h>
//
//using namespace std;
//
//#define DATASIZE 1048576 //2048 131072 262144 1048576���� �ȵƴµ� ���� �����÷ο� ���� ���� ���ִ� ����
//#define BLOCK_SIZE 2048  //1024 2048 2048 2048
////524288 1024 �����ɷ��� �׷��� �Ǳ��� 2097152 �� //80��,,
////1048576 1024 �ƹ�����ٷ���..
////1048576 2048 4194304�� 0.21�� 0.204�� ����
////1048576 4089 0.211�ʵ�� �켱 2048�̶� �����̴� ����
//
//void print_elapsed(clock_t start, clock_t stop) //�ð���¿�
//{
//    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
//    printf("CUDA ODD EVEN MERGE\n�ҿ�ð� : %.3fs\n", elapsed);
//}
//
//__global__ void oddevensort(int* input, unsigned int len, int i)
//{
//    //���� ���� ��ǥ
//    unsigned int tx = threadIdx.x;
//
//    //��ü �̹����� ��ǥ
//    unsigned int x = tx + blockDim.x * blockIdx.x;
//    //�̵��� �� �ӽ� ����
//    int temp;
//
//    //�ڷ��� ���̸�ŭ �����µ�, �ε���(i)�� ¦���̸� �������� ¦���ڸ��� �� ���� ���ڸ� ��.
//    //�ε����� Ȧ���̸� �������� Ȧ���ڸ��� �� ���� ���ڸ� ���ؼ� �����Ѵ�.
//    if (i % 2 == 0)
//    {
//        // ���̸� ���������ָ� ���� �����ִ� �ڸ��� �ִ� �ڷᰡ Ƣ��� �� ������ ����.
//        if (input[x] > input[x + 1] && x < len && x % 2 == 0)
//        {
//            temp = input[x + 1];
//            input[x + 1] = input[x];
//            input[x] = temp;
//        }
//    }
//    else
//    {
//        if (input[x] > input[x + 1] && x < len && x % 2 != 0)
//        {
//            temp = input[x + 1];
//            input[x + 1] = input[x];
//            input[x] = temp;
//        }
//    }
//    __syncthreads();
//}
//
//void array_print(int* arr, int length)
//{
//    int i;
//    for (i = 0; i < length; ++i) {
//        printf("%d ", arr[i]);
//    }
//    printf("\n");
//}
//
//void array_fill(int* arr, int length)
//{
//    srand((int)time(NULL));
//
//    for (int i = 0; i < length; ++i) {
//        arr[i] = rand() % 100;
//    }
//}
//
//__global__ void bitonic_sort_step(int* dev_values, int j, int k)
//{
//    unsigned int i, ixj; /* Sorting partners: i and ixj */
//    i = threadIdx.x + blockDim.x * blockIdx.x;
//    ixj = i ^ j;
//
//    /* The threads with the lowest ids sort the array. */
//    if ((ixj) > i) {
//        if ((i & k) == 0) {
//            /* Sort ascending */
//            if (dev_values[i] > dev_values[ixj]) {
//                /* exchange(i,ixj); */
//                int temp = dev_values[i];
//                dev_values[i] = dev_values[ixj];
//                dev_values[ixj] = temp;
//            }
//        }
//        if ((i & k) != 0) {
//            /* Sort descending */
//            if (dev_values[i] < dev_values[ixj]) {
//                /* exchange(i,ixj); */
//                int temp = dev_values[i];
//                dev_values[i] = dev_values[ixj];
//                dev_values[ixj] = temp;
//            }
//        }
//    }
//}
//
//clock_t start, stop = 0;
//int TestInput[DATASIZE], TestOutput[DATASIZE];
//
//int main(){
//    cout << "12163291\n������\n";
//
//    srand(time(NULL));
//
//    for (int i = 0; i < DATASIZE; i++) {
//        TestInput[i] = rand() % 100; //�����������ݴϴ�.
//    }
//
//    //device ����
//    int* devInput, * devOutput;
//    //�ϴ� ũ��� �ƴϱ�
//    unsigned int MemDataSize = DATASIZE * sizeof(int);
//
//    // device �ڸ� ����ݴϴ�.
//    cudaMalloc((void**)&devInput, MemDataSize);
//    cudaMalloc((void**)&devOutput, MemDataSize);
//    cudaMemset(devOutput, 0, MemDataSize);
//
//    // �ڸ� ������� ���縦 ���ݴϴ�.00000
//    cudaMemcpy(devInput, TestInput, MemDataSize, cudaMemcpyHostToDevice);
//
//    // block ũ�� �����մϴ�.
//    dim3    dimBlocksize(BLOCK_SIZE);
//    dim3    dimGridsize(ceil((DATASIZE - 1) / (float)BLOCK_SIZE) + 1);
//    // �ϴ� Max���� min���� �˾ƾ� �մϴ�.
//    // ���� �����ϴ� cpu���� �ð��ݴϴ�.
//    
//    start = clock();
//    for (int i = 0; i < DATASIZE; i++) {
//        oddevensort << < dimGridsize, dimBlocksize >> > (devInput, DATASIZE, i);
//    }
//    stop = clock();
//
//    // ����� testoutput�� �����մϴ�.
//    cudaMemcpy(TestOutput, devInput, MemDataSize, cudaMemcpyDeviceToHost);
//
// /*   for (int i = 0; i < DATASIZE; i++) {
//        cout << TestOutput[i] << ", ";  //��� Ȯ�ο�
//        if ((i + 1) % 16 == 0)  {
//            cout << endl;
//        }
//    }*/
//    cout << endl << "�迭 ũ�� : " << sizeof(TestOutput) << endl << endl;
//   print_elapsed(start, stop); //�ɸ��ð� ������ݴϴ�.
//
//   //���� GPU�� ������ �ڸ� free.
//   //malloc - free�� �׻� ¦������. 
//   //�׻������ �մϴ�. 
//    cudaFree(devInput);
//    cudaFree(devOutput);
//
//    return 0;
//}
//
////https://github.com/ptjoker95/OddEvenSort-with-CUDA/blob/master/oddevensort.cu
////��ó�� ���� �����ϴ�.