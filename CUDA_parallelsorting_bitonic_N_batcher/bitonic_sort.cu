#include "hip/hip_runtime.h"
//12163291 ������ �˰��򼳰� HW1

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 256 //16, 128, 128, 256
#define BLOCKS 16384 //512, 4096, 8192, 16384
#define NUM_VALS THREADS*BLOCKS //���ĵǴ� element�� ����

void print_elapsed(clock_t start, clock_t stop) //�ɸ� �ð� ��� �Լ�
{
    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("\nCUDA BITONIC MERGE\n�ҿ�ð� : %.3fs\n", elapsed);
}

void array_print(int* arr, int length) //���ĵ� ����� ���
{
    int i;
    for (i = 0; i < length; ++i) {
        printf("%d ", arr[i]);
        if ((i + 1) % 16 == 0){ //16�� ���� endl
            cout << endl;
        }
    }
    printf("\n");
}

void array_fill(int* arr, int length) //NUM_VALS ũ���� �迭�� ���� ������ ä���ֱ�
{
    srand((int)time(NULL));

    for (int i = 0; i < length; ++i) {
        arr[i] = rand() % 100;
    }
}

__global__ void bitonic_sort_step(int* dev_values, int j, int k)
{
    unsigned int i, ixj; // Sorting ��Ʈ�ʵ��� : i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    // The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort �������� */
            if (dev_values[i] > dev_values[ixj]) {
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0) {
            /* Sort �������� */
            if (dev_values[i] < dev_values[ixj]) {
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(int* values)
{
    int* dev_values;
    size_t size = NUM_VALS * sizeof(int); //����� size_t�� 

    hipMalloc((void**)&dev_values, size); //malloc = �Ҵ��Ѵ� 
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);    /* �������� �ռ� ������ �־��� Number of ���� ���� */
    dim3 threads(THREADS, 1);  /* Number of ������ ���� */

    int j, k;
    /* �߿��� �κ� */
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        /* Minor */
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step << <blocks, threads >> > (dev_values, j, k); //�ռ� �����ص� ���� sort����
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost); //values�鿡 ���ĵ� �� cp���ִ� ��
    hipFree(dev_values); //Ǯ���ֱ�
}

int main(void)
{
    cout << "12163291\n������\n";
    clock_t start, stop;

    int* values = (int*)malloc(NUM_VALS * sizeof(int));
    array_fill(values, NUM_VALS);

    start = clock();
    bitonic_sort(values); /* Inplace */
    stop = clock();
    //array_print(values, NUM_VALS);

    print_elapsed(start, stop);
}

//https://github.com/snowacat/cuda-bitonic-sorting/blob/master/bitonic.cu
//��ó�� ���� �����ϴ�.